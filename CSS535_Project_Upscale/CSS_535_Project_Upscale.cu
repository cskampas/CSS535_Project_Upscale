﻿#if _MSC_VER
#include "hip/hip_runtime.h"
#include ""
#endif

#include "bitmap.h"

#include <iostream>

using namespace std;

int main()
{
    cout << "Hello, World!" << endl;
    Bitmap* b = new Bitmap();
    b->readFromFile("C:\\Users\\cskam\\source\repos\\CSS535_Project_Upscale\\CSS535_Project_Upscale\\TestContent\\Test1.bmp");
    b->writeToFile("TestContent\\Test2.bmp");
    return 0;
}
