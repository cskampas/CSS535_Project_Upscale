#include "hip/hip_runtime.h"
﻿#if _MSC_VER
#include "hip/hip_runtime.h"
#include ""
#endif

#include "bitmap.h"

#include <iostream>

using namespace std;

void print_matrix(unsigned char* matrix, unsigned short width, unsigned short height, int pad){
	for (int y = 0; y < height; ++y)
	{
		for (int x = 0; x < width; ++x)
		{
			unsigned char* pixel = matrix + (y * width * 3 + x * 3 + y * pad);

			if (pixel[0] == 255 && pixel[1] == 255 && pixel[2] == 255)
				cout << " ";
			else if (pixel[0] == 0 && pixel[1] == 0 && pixel[2] == 0)
				cout << "X";
			else if (pixel[2] == 255)
				cout << "R";
			else if (pixel[1] == 255)
				cout << "G";
			else if (pixel[0] == 255)
				cout << "B";
			else
				cout << "?";
		}
		cout << endl;
	}
}

__global__ void NearestNeighbor(
	unsigned char* source,
	unsigned short oWidth,
	unsigned short oHeight,
	unsigned char oPad,
	unsigned char* dest,
	unsigned short nWidth,
	unsigned short nHeight,
	unsigned char nPad)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (col >= nWidth || row >= nHeight)
	{
		return;
	}

	int index = ((col + row * nWidth) * 3) + row * nPad;

	int oCol = (int)(((float)col / (float)nWidth) * oWidth + 0.5f);
	int oRow = (int)(((float)row / (float)nHeight) * oHeight + 0.5f);

	if (oCol < 0)
	{
		oCol = 0;
	}
	if (oCol >= oHeight)
	{
		oCol = oHeight - 1;
	}
	if (oRow < 0)
	{
		oRow = 0;
	}
	if (oRow >= oHeight)
	{
		oRow = oHeight - 1;
	}

	int oIndex = ((oCol + oRow * oWidth) * 3) + oRow * oPad;

	dest[index] = source[oIndex];
	dest[index + 1] = source[oIndex + 1];
	dest[index + 2] = source[oIndex + 2];
}

__global__ void Bilinear(
	unsigned char* source,
	unsigned short oWidth,
	unsigned short oHeight,
	unsigned char oPad,
	unsigned char* dest,
	unsigned short nWidth,
	unsigned short nHeight,
	unsigned char nPad)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	if (col >= nWidth || row >= nHeight)
	{
		return;
	}

	int index = ((col + row * nWidth) * 3) + row * nPad;

	// Find left and right pixel from row above and row below
	// "Top" and "Left" here means towards 0, regardless of the reality of the image format

	float sourceRelativeRow = (float)row / (float)nHeight;
	float sourceRelativeCol = (float)col / (float)nWidth;

	// int oCol = (int)(sourceRelativeCol * oWidth + 0.5f);
	// int oRow = (int)(sourceRelativeRow * oHeight + 0.5f);
	// int oIndex = ((oCol + oRow * oWidth) * 3) + oRow * oPad;

	int oRowTop = (int)(sourceRelativeRow * oHeight);
	int oRowBot = (int)(sourceRelativeRow * oHeight) + 1;
	int oColLeft = (int)(sourceRelativeCol * oWidth);
	int oColRight = (int)(sourceRelativeCol * oWidth) + 1;
	/*
	if (oColLeft < 0)
	{
		oColLeft = 0;
	}
	if (oColRight >= oHeight)
	{
		oColRight = oHeight - 1;
	}
	if (oRowTop < 0)
	{
		oRowTop = 0;
	}
	if (oRowBot >= oHeight)
	{
		oRowBot = oHeight - 1;
	}*/

	// Bilinear calculation
	unsigned char topLeft[3];
	unsigned char topRight[3];
	unsigned char botLeft[3];
	unsigned char botRight[3];
	int oColLeftSample = oColLeft;
	int oColRightSample = oColRight;
	int oRowTopSample = oRowTop;
	int oRowBotSample = oRowBot;

	if (oColLeft < 0)
	{
		oColLeftSample = 0;
	}
	if (oColRight >= oWidth)
	{
		oColRightSample = oWidth - 1;
	}
	if (oRowTop < 0)
	{
		oRowTopSample = 0;
	}
	if (oRowBot >= oHeight)
	{
		oRowBotSample = oHeight - 1;
	}
	int oIndexTL = ((oColLeftSample + oRowTopSample * oWidth) * 3) + oRowTopSample * oPad;
	int oIndexTR = ((oColRightSample + oRowTopSample * oWidth) * 3) + oRowTopSample * oPad;
	int oIndexBL = ((oColLeftSample + oRowBotSample * oWidth) * 3) + oRowBotSample * oPad;
	int oIndexBR = ((oColRightSample + oRowBotSample * oWidth) * 3) + oRowBotSample * oPad;
	unsigned char TL[3];
	unsigned char TR[3];
	unsigned char BL[3];
	unsigned char BR[3];

	float leftLinearFactor = sourceRelativeCol - oColLeft;
	float rightLinearFactor = oColRight - sourceRelativeCol;
	float topLinearFactor = sourceRelativeRow - oRowTop;
	float botLinearFactor = oRowBot - sourceRelativeRow;

	for (int i = 0; i < 3; ++i)
	{
		TL[i] = source[oIndexTL + i];
		TR[i] = source[oIndexTR + i];
		BL[i] = source[oIndexBL + i];
		BR[i] = source[oIndexBR + i];

		float top = leftLinearFactor * TL[i] + rightLinearFactor * TR[i];
		float bot = leftLinearFactor * BL[i] + rightLinearFactor * BR[i];
		float result = topLinearFactor * top + botLinearFactor * bot;
		dest[index + i] = static_cast<unsigned char>(oRowBot * 128);
	}

	/*
	float bTop = leftLinearFactor * TL[0] + rightLinearFactor * TR[0];
	float gTop = leftLinearFactor * TL[1] + rightLinearFactor * TR[1];
	float rTop = leftLinearFactor * TL[2] + rightLinearFactor * TR[2];

	float bBot = leftLinearFactor * BL[0] + rightLinearFactor * BR[0];
	float gBot = leftLinearFactor * BL[1] + rightLinearFactor * BR[1];
	float rBot = leftLinearFactor * BL[2] + rightLinearFactor * BR[2];

	float b = topLinearFactor * bTop + botLinearFactor * bBot;
	float g = topLinearFactor * gTop + botLinearFactor * gBot;
	float r = topLinearFactor * rTop + botLinearFactor * rBot;


	dest[index] = (unsigned char)bTop;
	dest[index + 1] = (unsigned char)gTop;
	dest[index + 2] = (unsigned char)rTop;
	*/
	/*
	dest[index] = (unsigned char)b;
	dest[index + 1] = (unsigned char)g;
	dest[index + 2] = (unsigned char)r;
	*/
	/*

	int oCol = (int)(((float)col / (float)nWidth) * oWidth + 0.5f);
	int oRow = (int)(((float)row / (float)nHeight) * oHeight + 0.5f);

	if (oCol < 0)
	{
		oCol = 0;
	}
	if (oCol >= oHeight)
	{
		oCol = oHeight - 1;
	}
	if (oRow < 0)
	{
		oRow = 0;
	}
	if (oRow >= oHeight)
	{
		oRow = oHeight - 1;
	}

	int oIndex = ((oCol + oRow * oWidth) * 3) + oRow * oPad;

	dest[index] = source[oIndex];
	dest[index + 1] = source[oIndex + 1];
	dest[index + 2] = source[oIndex + 2];
	*/
}

void BilinearCPUKernel(
	unsigned int row,
	unsigned int col,
	unsigned char* source,
	unsigned short oWidth,
	unsigned short oHeight,
	unsigned char oPad,
	unsigned char* dest,
	unsigned short nWidth,
	unsigned short nHeight,
	unsigned char nPad)
{
if (col >= nWidth || row >= nHeight)
{
	return;
}

int index = ((col + row * nWidth) * 3) + row * nPad;

// Find left and right pixel from row above and row below
// "Top" and "Left" here means towards 0, regardless of the reality of the image format

float sourceRelativeRow = (float)row / (float)nHeight;
float sourceRelativeCol = (float)col / (float)nWidth;

// int oCol = (int)(sourceRelativeCol * oWidth + 0.5f);
// int oRow = (int)(sourceRelativeRow * oHeight + 0.5f);
// int oIndex = ((oCol + oRow * oWidth) * 3) + oRow * oPad;

int oRowTop = (int)(sourceRelativeRow * oHeight);
int oRowBot = (int)(sourceRelativeRow * oHeight) + 1;
int oColLeft = (int)(sourceRelativeCol * oWidth);
int oColRight = (int)(sourceRelativeCol * oWidth) + 1;
/*
if (oColLeft < 0)
{
	oColLeft = 0;
}
if (oColRight >= oHeight)
{
	oColRight = oHeight - 1;
}
if (oRowTop < 0)
{
	oRowTop = 0;
}
if (oRowBot >= oHeight)
{
	oRowBot = oHeight - 1;
}*/

// Bilinear calculation
unsigned char topLeft[3];
unsigned char topRight[3];
unsigned char botLeft[3];
unsigned char botRight[3];
int oColLeftSample = oColLeft;
int oColRightSample = oColRight;
int oRowTopSample = oRowTop;
int oRowBotSample = oRowBot;

if (oColLeft < 0)
{
	oColLeftSample = 0;
}
if (oColRight >= oWidth)
{
	oColRightSample = oWidth - 1;
}
if (oRowTop < 0)
{
	oRowTopSample = 0;
}
if (oRowBot >= oHeight)
{
	oRowBotSample = oHeight - 1;
}
int oIndexTL = ((oColLeftSample + oRowTopSample * oWidth) * 3) + oRowTopSample * oPad;
int oIndexTR = ((oColRightSample + oRowTopSample * oWidth) * 3) + oRowTopSample * oPad;
int oIndexBL = ((oColLeftSample + oRowBotSample * oWidth) * 3) + oRowBotSample * oPad;
int oIndexBR = ((oColRightSample + oRowBotSample * oWidth) * 3) + oRowBotSample * oPad;
unsigned char TL[3];
unsigned char TR[3];
unsigned char BL[3];
unsigned char BR[3];

float leftLinearFactor = sourceRelativeCol - oColLeft;
float rightLinearFactor = oColRight - sourceRelativeCol;
float topLinearFactor = sourceRelativeRow - oRowTop;
float botLinearFactor = oRowBot - sourceRelativeRow;

for (int i = 0; i < 3; ++i)
{
	TL[i] = source[oIndexTL + i];
	TR[i] = source[oIndexTR + i];
	BL[i] = source[oIndexBL + i];
	BR[i] = source[oIndexBR + i];

	float top = leftLinearFactor * TL[i] + rightLinearFactor * TR[i];
	float bot = leftLinearFactor * BL[i] + rightLinearFactor * BR[i];
	float result = topLinearFactor * top + botLinearFactor * bot;
	dest[index + i] = static_cast<unsigned char>(result);
}

/*
float bTop = leftLinearFactor * TL[0] + rightLinearFactor * TR[0];
float gTop = leftLinearFactor * TL[1] + rightLinearFactor * TR[1];
float rTop = leftLinearFactor * TL[2] + rightLinearFactor * TR[2];

float bBot = leftLinearFactor * BL[0] + rightLinearFactor * BR[0];
float gBot = leftLinearFactor * BL[1] + rightLinearFactor * BR[1];
float rBot = leftLinearFactor * BL[2] + rightLinearFactor * BR[2];

float b = topLinearFactor * bTop + botLinearFactor * bBot;
float g = topLinearFactor * gTop + botLinearFactor * gBot;
float r = topLinearFactor * rTop + botLinearFactor * rBot;


dest[index] = (unsigned char)bTop;
dest[index + 1] = (unsigned char)gTop;
dest[index + 2] = (unsigned char)rTop;
*/
/*
dest[index] = (unsigned char)b;
dest[index + 1] = (unsigned char)g;
dest[index + 2] = (unsigned char)r;
*/
/*

int oCol = (int)(((float)col / (float)nWidth) * oWidth + 0.5f);
int oRow = (int)(((float)row / (float)nHeight) * oHeight + 0.5f);

if (oCol < 0)
{
	oCol = 0;
}
if (oCol >= oHeight)
{
	oCol = oHeight - 1;
}
if (oRow < 0)
{
	oRow = 0;
}
if (oRow >= oHeight)
{
	oRow = oHeight - 1;
}

int oIndex = ((oCol + oRow * oWidth) * 3) + oRow * oPad;

dest[index] = source[oIndex];
dest[index + 1] = source[oIndex + 1];
dest[index + 2] = source[oIndex + 2];
*/
}

void BilinearCPU(
	unsigned int blockIdxX,
	unsigned int blockIdxY,
	unsigned int threadIdxX,
	unsigned int threadIdxY,
	unsigned char* source,
	unsigned short oWidth,
	unsigned short oHeight,
	unsigned char oPad,
	unsigned char* dest,
	unsigned short nWidth,
	unsigned short nHeight,
	unsigned char nPad)
{
	for (unsigned int gX = 0; gX < blockIdxX; ++gX)
	{
		for (unsigned int gY = 0; gY < blockIdxY; ++gY)
		{
			for (unsigned int bX = 0; bX < threadIdxX; ++bX)
			{
				for (unsigned int bY = 0; bY < threadIdxY; ++bY)
				{
					int col = bX + gX * 32;
					int row = bY + gY * 32;
					BilinearCPUKernel(row, col, source, oWidth, oHeight, oPad, dest, nWidth, nHeight, nPad);
				}
			}
		}
	}
}

#define BLOCK_SIZE 2
__global__ void CopyImage(unsigned char* a, unsigned char* b, unsigned short width, unsigned short height, int pad) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	int index = ((col + row * width) * 3) + row * pad;

	if (row < height && col < width) {
		b[index] = a[index];
        b[index + 1] = a[index + 1];
        b[index + 2] = a[index + 2];
	}
}

void NearestNeighbor(Bitmap* source, Bitmap* dest)
{
	const int NearestNeighborBlockSize = 32;
	dest->init();

	unsigned short oW = source->width;
	unsigned short oH = source->height;
	unsigned char oP = source->padSize();
	unsigned short nW = dest->width;
	unsigned short nH = dest->height;
	unsigned char nP = dest->padSize();

	unsigned char* original_image, * upscaled_image;
	unsigned char* original_image_device, * upscaled_image_device;

	int size_matrix = source->imageDataSize();
	int size_dest = dest->imageDataSize();
	original_image = source->imageData;
	upscaled_image = dest->imageData;

	hipMalloc((void**)&original_image_device, size_matrix);
	hipMalloc((void**)&upscaled_image_device, size_dest);

	hipMemcpy(original_image_device, original_image, size_matrix, hipMemcpyHostToDevice);

	dim3 dimBlock(NearestNeighborBlockSize, NearestNeighborBlockSize);
	dim3 dimGrid((nW / dimBlock.x) + 1, (nH / dimBlock.y) + 1);

	NearestNeighbor <<<dimGrid, dimBlock>>>(original_image_device, oW, oH, oP, upscaled_image_device, nW, nH, nP);

	hipMemcpy(upscaled_image, upscaled_image_device, size_dest, hipMemcpyDeviceToHost);

	hipFree(original_image_device);
	hipFree(upscaled_image_device);
}

void Bilinear(Bitmap* source, Bitmap* dest)
{
	const int BilinearBlockSize = 32;
	dest->init();

	unsigned short oW = source->width;
	unsigned short oH = source->height;
	unsigned char oP = source->padSize();
	unsigned short nW = dest->width;
	unsigned short nH = dest->height;
	unsigned char nP = dest->padSize();

	unsigned char* original_image, * upscaled_image;
	unsigned char* original_image_device, * upscaled_image_device;

	int size_matrix = source->imageDataSize();
	int size_dest = dest->imageDataSize();
	original_image = source->imageData;
	upscaled_image = dest->imageData;

	hipMalloc((void**)&original_image_device, size_matrix);
	hipMalloc((void**)&upscaled_image_device, size_dest);

	hipMemcpy(original_image_device, original_image, size_matrix, hipMemcpyHostToDevice);

	dim3 dimBlock(BilinearBlockSize, BilinearBlockSize);
	dim3 dimGrid((nW / dimBlock.x) + 1, (nH / dimBlock.y) + 1);

	Bilinear<<<dimGrid, dimBlock>>>(original_image_device, oW, oH, oP, upscaled_image_device, nW, nH, nP);

	hipMemcpy(upscaled_image, upscaled_image_device, size_dest, hipMemcpyDeviceToHost);

	hipFree(original_image_device);
	hipFree(upscaled_image_device);
}

int main()
{
    Bitmap* baseImage = new Bitmap();
	Bitmap* nearestNeighborImage = new Bitmap();
	Bitmap* bilinearImage = new Bitmap();
	nearestNeighborImage->width = 295;
	nearestNeighborImage->height = 295;
	bilinearImage->width = 1005;
	bilinearImage->height = 1005;
	baseImage->readFromFile("TestContent/Test1.bmp");
	NearestNeighbor(baseImage, nearestNeighborImage);
	const int BilinearBlockSize = 32;
	bilinearImage->init();

	unsigned short oW = baseImage->width;
	unsigned short oH = baseImage->height;
	unsigned char oP = baseImage->padSize();
	unsigned short nW = bilinearImage->width;
	unsigned short nH = bilinearImage->height;
	unsigned char nP = bilinearImage->padSize();

	unsigned char* original_image, * upscaled_image;
	unsigned char* original_image_device, * upscaled_image_device;

	int size_matrix = baseImage->imageDataSize();
	int size_dest = bilinearImage->imageDataSize();
	original_image = baseImage->imageData;
	upscaled_image = bilinearImage->imageData;

	//dim3 dimBlock(BilinearBlockSize, BilinearBlockSize);
	//dim3 dimGrid((nW / dimBlock.x) + 1, (nH / dimBlock.y) + 1);
	//uint3 dB = dimBlock;
	//uint3 dG = dimGrid;

	BilinearCPU(32, 32, (nW / 32) + 1, (nH / 32) + 1, original_image, oW, oH, oP, upscaled_image, nW, nH, nP);

	//Bilinear(baseImage, bilinearImage);
	nearestNeighborImage->writeToFile("TestContent/Test1NearestNeighbor.bmp");
	bilinearImage->writeToFile("TestContent/Test1Bilinear.bmp");

	return 0;

	// result->init();

	// int oW = b->width;
	// int oH = b->height;
	// int oP = b->padSize();
	// int nW = result->width;
	// int nH = result->height;
	// int nP = result->padSize();

    // b->writeToFile("TestContent/Test2.bmp");
	
    // unsigned short width = b->width;
	// unsigned short height = b->height;
    
	// unsigned char *original_image, *upscaled_image;
    // unsigned char *original_image_device, *upscaled_image_device;
    
	// int size_matrix = b->imageDataSize();
	// int size_dest = result->imageDataSize();
	// original_image = b->imageData;
	// upscaled_image = result->imageData;

    // upscaled_image = (unsigned char*)malloc(size_matrix);

    // hipMalloc((void**)&original_image_device, size_matrix);
    // hipMalloc((void**)&upscaled_image_device, size_dest);

    // hipMemcpy(original_image_device, original_image, size_matrix, hipMemcpyHostToDevice);

    //dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 dimGrid(width / dimBlock.x, height / dimBlock.y);
    //CopyImage <<<dimGrid, dimBlock >>>(original_image_device, upscaled_image_device, width, height, b->padSize());

	// dim3 dimBlock(32, 32);
	// dim3 dimGrid((nW / dimBlock.x) + 1, (nH / dimBlock.y) + 1);
	// NearestNeighbor <<<dimGrid, dimBlock >>> (original_image_device, oW, oH, oP, upscaled_image_device, nW, nH, nP);

    // hipMemcpy(upscaled_image, upscaled_image_device, size_dest, hipMemcpyDeviceToHost);
	/*
    cout << "ORIGINAL" << endl;
    print_matrix(original_image, width, height, b->padSize());
    cout << "COPY" << endl;
    print_matrix(upscaled_image, width, height, b->padSize());
	*/

	// result->writeToFile("TestContent/Test1NearestNeighbor.bmp");

    // hipFree(original_image_device);
    // hipFree(upscaled_image_device);

	// ------------------------

    // return 0;
}

// Kernel test code
/*
unsigned char r = index % 256;
unsigned char g = (index / 256) % 256;
unsigned char b = (index / 65536) % 256;

dest[index] = b;
dest[index + 1] = g;
dest[index + 2] = r;
*/

/*
int oCol = (int)(((float)col / (float)nWidth) * oWidth + 0.5f);
int oRow = (int)(((float)row / (float)nHeight) * oHeight + 0.5f);

int oIndex = ((oCol + oRow * nWidth) * 3) + oRow * oPad;

dest[index] = source[oIndex];
dest[index + 1] = source[oIndex + 1];
dest[index + 2] = source[oIndex + 2];
*/

// int oCol = col / 4;
// int oRow = row / 4;
